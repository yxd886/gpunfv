#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "gpu_ips.cuh"

using namespace std;

#define THREADPERBLOCK	256
#define SHARE_MEM_SIZE  256

__global__ void testKernel(char *s) {
	int i = threadIdx.x;
	
	memcpy(s + i + 5, s + i, 1);
	//assert(1);
}

bool test_cudaHostAlloc() {
	bool res = true;
	char *dptr1, *hptr1;

	// First way: using hipHostAlloc()
	// Alloc host page-locked memory
	checkCudaErrors(hipHostAlloc(&hptr1, 10, hipHostMallocMapped));

	// Get corresponding device pointer
	checkCudaErrors(hipHostGetDevicePointer(&dptr1, hptr1, 0));

	// Initialize this memory
	for(int i = 0; i < 5; i++)
		hptr1[i] = i;
	for(int i = 5; i < 10; i++)
		hptr1[i] = 0;

	// Test kernel
	testKernel<<<1, 5>>>(dptr1);
	hipDeviceSynchronize();

	// Check result
	for(int i = 0; i < 10; i++){
		printf("s[%d]: %d\n", i, hptr1[i]);
		res = (hptr1[i] == i % 5) ? res : false;
	}

	// Free memory
	checkCudaErrors(hipHostFree(hptr1));

	return res;
}

bool test_cudaHostRegister() {
	bool res = true;
	char *dptr1, *hptr1;

	// Second way: using hipHostRegister()
	// Alloc host memory
	hptr1 = new char[10];
	//assert(hptr1);

	// Page-lock host memory
	hipHostRegister(hptr1, 10, hipHostRegisterMapped);

	// Get corresponding device pointer
	checkCudaErrors(hipHostGetDevicePointer(&dptr1, hptr1, 0));

	// Initialize this memory
	for(int i = 0; i < 5; i++)
		hptr1[i] = i;
	for(int i = 5; i < 10; i++)
		hptr1[i] = 0;

	// Test kernel
	if(hipDeviceAttributeCanUseHostPointerForRegisteredMem != 0){
		printf("Can directly use host pointer to substitute device pointer on this machine.\n");
		testKernel<<<1, 5>>>(hptr1);
	}
	else{
		printf("This machine does not support substituting host pointer for device pointer.\n");
		testKernel<<<1, 5>>>(dptr1);
	}
	hipDeviceSynchronize();

	// Check result
	for(int i = 0; i < 10; i++){
		printf("s[%d]: %d\n", i, hptr1[i]);
		res = (hptr1[i] == i % 5) ? res : false;
	}

	// Free memory
	hipHostUnregister(hptr1);
	delete hptr1;

	return res;
}

void start_test() {
	// Enable memory mapping
	hipSetDeviceFlags(hipDeviceMapHost);

	if(test_cudaHostAlloc())
		printf("hipHostAlloc(): PASS\n");
	else
		printf("hipHostAlloc(): ERROR\n");

	if(test_cudaHostRegister())
		printf("hipHostRegister(): PASS\n");
	else
		printf("hipHostRegister(): ERROR\n");
}

__global__ void gpu_nf_logic(char** pkt_batch, char **state_batch, char *extra_info, int flowDim, int nflows) {
	//printf("in gpu_nf_logic\n");
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	if(id >= nflows) return ;

	// Get start address
	//printf("sizeof bool: %d\n",sizeof(bool));
	char**pkts =pkt_batch + id * flowDim;
	//struct ips_flow_state* state_ptr=(struct ips_flow_state*)state_batch;

	//printf("pkt_batch = %x\n", pkt_batch);

	
	//printf("flowDim = %d, id = %d, pkts = %p, pkts[0] = %p\n", flowDim, id, pkts, pkts[0]);
	// For every packet for this flow in this batch
	for(int i = 0; i < flowDim; i++) {
	//printf("id = %d, i = %d, pkts[i] = %p\n", id, i, pkts[i]);	
		if(pkts[i] == NULL) break;
 //printf("gpu_nf_logic(): state->_dfa_id = %d\n", ((struct ips_flow_state *)state_batch[id])->_dfa_id);
		//gpu_nf_logic_impl(pkts[i], state_batch[id]);

		//ips_detect((char*)pkts[i].pkt, &state_ptr[id], (struct gpu_IPS *)extra_info);
		ips_detect(pkts[i], (struct ips_flow_state *)state_batch[id], (struct gpu_IPS *)extra_info);

//	printf("id = %d, end\n", id);	
	}
	//printf("GPU: gpu_states[%d].dfa_id: %d\n",id,state_ptr[id]._dfa_id);
}

void gpu_launch(char **pkt_batch, char **state_batch, char *extra_info, int flowDim, int nflows,hipStream_t stream) {
	// Calculate block amounts
	assert(nflows > 0);
	int nblocks = (nflows + THREADPERBLOCK - 1) / THREADPERBLOCK;
//printf("nblocks = %d, nthread = %d, nflows = %d\n", nblocks, THREADPERBLOCK, nflows);
	//gpu_nf_logic<<<nblocks, THREADPERBLOCK, SHARE_MEM_SIZE, stream>>>(pkt_batch, state_batch, extra_info, flowDim, nflows);
	gpu_nf_logic<<<1, 1, SHARE_MEM_SIZE, stream>>>(pkt_batch, state_batch, extra_info, flowDim, nflows);
}

void gpu_sync(hipStream_t stream) {
	checkCudaErrors(hipStreamSynchronize(stream));
}

void gpu_mem_map(void *ptr, size_t size) {
	checkCudaErrors(hipHostRegister(ptr, size, hipHostRegisterMapped));
}

void gpu_mem_unmap(void *ptr) {
	checkCudaErrors(hipHostUnregister(ptr));
}

void gpu_malloc(void** devPtr, size_t size){
	checkCudaErrors(hipMalloc(devPtr, size));
}


void gpu_memcpy_async_h2d(void* dst, const void*src, size_t count ,hipStream_t stream=0){
	checkCudaErrors(hipMemcpyAsync(dst,src,count,hipMemcpyHostToDevice,stream));
}

void gpu_memcpy_async_d2h(void* dst, const void*src, size_t count, hipStream_t stream=0 ){
	checkCudaErrors(hipMemcpyAsync(dst,src,count,hipMemcpyDeviceToHost,stream));
	
}


void gpu_memset_async(void * devPtr, int value, size_t count, hipStream_t stream = 0){

	checkCudaErrors(hipMemsetAsync(devPtr,value,count,stream))	;
}

void gpu_free(void* devPtr){
	checkCudaErrors(hipFree(devPtr));
}




void create_stream(hipStream_t* stream_ptr){

	checkCudaErrors(hipStreamCreate(stream_ptr));
}

void destory_stream(hipStream_t stream){

checkCudaErrors(hipStreamDestroy(stream));
}