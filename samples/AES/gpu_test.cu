#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "../include/gpu_interface.hh"
#include "nf.cuh"
//#include "packet_parser.cuh"

using namespace std;

#define THREADPERBLOCK	256
#define SHARE_MEM_SIZE  512
#define MAX_PKT_SIZE 	64

struct PKT{
	char pkt[MAX_PKT_SIZE];
};

__device__ size_t align_access(size_t a,size_t radix){
	return ((a+radix-1)/radix)*radix;
}

__global__ void gpu_nf_logic(char *pkt_batch, char *state_batch, char *extra_info, int flowDim, int nflows) {

	__shared__ nf_flow_state gpu_nf_flow_state[32];

	
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	if(id >= nflows) return ;

	Infos *info = (Infos *)extra_info;
	char *messages =pkt_batch + id * flowDim;
	nf_flow_state *states = (nf_flow_state *)state_batch;

	// Copy state to shared memory
	gpu_nf_flow_state[id%32] = states[id];
	size_t len = align_access(*((size_t*)messages)+sizeof(size_t),sizeof(size_t));
	//printf("gpuside len:%d\n",len);
	size_t total_len = 0;
	while(len) {
		NF::nf_logic(messages, &gpu_nf_flow_state[id % 32], extra_info);
		//printf("gpu process messages\n");
		messages+=len;
		total_len+=len;
		if(total_len>=flowDim){
			break;
		}
		len = align_access(*((size_t*)messages+sizeof(size_t)),sizeof(size_t));
	}

	// Copy state back from shared memory
	states[id]= gpu_nf_flow_state[id % 32];

}

void gpu_launch(char *pkt_batch, char *state_batch, char *extra_info, int flowDim, int nflows,hipStream_t stream) {
	// Calculate block amounts
	assert(nflows > 0);
	int nblocks = (nflows + THREADPERBLOCK - 1) / THREADPERBLOCK;
//printf("-----------------------nblocks = %d, nthread = %d, nflows = %d\n", nblocks, THREADPERBLOCK, nflows);
	gpu_nf_logic<<<nblocks, THREADPERBLOCK, SHARE_MEM_SIZE, stream>>>(pkt_batch, state_batch, extra_info, flowDim, nflows);
	//gpu_nf_logic<<<1, 1, SHARE_MEM_SIZE, stream>>>(pkt_batch, state_batch, extra_info, flowDim, nflows);
}

void *gpu_malloc_set(size_t size, void *data) {
	void *devPtr;

	gpu_malloc(&devPtr, size);
	checkCudaErrors(hipMemcpy(devPtr, data, size, hipMemcpyHostToDevice));

	return devPtr;
}

void gpu_sync(hipStream_t stream) {
	checkCudaErrors(hipStreamSynchronize(stream));
}

void gpu_mem_map(void *ptr, size_t size) {
	checkCudaErrors(hipHostRegister(ptr, size, hipHostRegisterMapped));
}

void gpu_mem_unmap(void *ptr) {
	checkCudaErrors(hipHostUnregister(ptr));
}

void gpu_malloc(void** devPtr, size_t size){
	checkCudaErrors(hipMalloc(devPtr, size));
}

void gpu_malloc_host(void** devPtr, size_t size){
	checkCudaErrors(hipHostMalloc(devPtr, size));
}

void gpu_memcpy_async_h2d(void* dst, const void*src, size_t count ,hipStream_t stream){
	checkCudaErrors(hipMemcpyAsync(dst,src,count,hipMemcpyHostToDevice,stream));
}

void gpu_memcpy_async_d2h(void* dst, const void*src, size_t count, hipStream_t stream){
	checkCudaErrors(hipMemcpyAsync(dst,src,count,hipMemcpyDeviceToHost,stream));
}


void gpu_memset_async(void * devPtr, int value, size_t count, hipStream_t stream){
	checkCudaErrors(hipMemsetAsync(devPtr,value,count,stream))	;
}

void gpu_free(void* devPtr){
	checkCudaErrors(hipFree(devPtr));
}

void gpu_free_host(void* devPtr){
	checkCudaErrors(hipHostFree(devPtr));
}

void create_stream(hipStream_t* stream_ptr){

	checkCudaErrors(hipStreamCreateWithFlags(stream_ptr,hipStreamNonBlocking));
}

void destory_stream(hipStream_t stream){
	checkCudaErrors(hipStreamDestroy(stream));
}

void create_event(hipEvent_t* event_ptr){
	checkCudaErrors(hipEventCreateWithFlags(event_ptr,hipEventDisableTiming));
}
